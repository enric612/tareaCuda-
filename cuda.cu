#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <deque>
#include <stack>
#include <FreeImage.h>
#include <limits.h>
#include <stdlib.h>
#include <cstdlib>
#include <cstdio>

//Funció per a determinar errors
static void HandleError( hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

// MACRO per utilitzar la funció anterior
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void cal_cuda_grises(BYTE * c_greybits,  BYTE * c_bits, int width, int scan_width, int height)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if(i<height && j < width)
    {
        float r = (float) (c_bits[i * scan_width + j * 4 + 0]);
        float g = (float) (c_bits[i * scan_width + j * 4 + 1]);
        float b = (float) (c_bits[i * scan_width + j * 4 + 2]);
        float valor = 0.2126 * r + 0.7152 *g + 0.0722 * b;
        c_greybits[i * width + j] = (BYTE) (valor);
    }
}

extern "C" void cuda_grises(BYTE * greybits, BYTE * greybits_PG, BYTE * bits, int height, int width, int scan_width)
{

    BYTE *c_greybits, *c_bits;

    HANDLE_ERROR(hipMalloc((void**)&c_bits, height*scan_width));
    HANDLE_ERROR(hipMalloc((void**)&c_greybits, height*width));

    HANDLE_ERROR(hipMemcpy(c_bits, bits, height*scan_width, hipMemcpyHostToDevice));
    

    int x = static_cast<int>(ceilf(static_cast<float>(width) / 32));
    int y = static_cast<int>(ceilf(static_cast<float>(height) / 32));
    const dim3 red(x,y,1);
    const dim3 bloque(32,32,1);
    cal_cuda_grises<<<red, bloque>>>(c_greybits, c_bits, width, scan_width, height);
  
    HANDLE_ERROR(hipMemcpy(greybits,c_greybits,height*width,hipMemcpyDeviceToHost));
    
    hipFree(c_bits);
    hipFree(c_greybits);

}
