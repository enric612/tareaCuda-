#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <deque>
#include <stack>
#include <FreeImage.h>
#include <limits.h>
#include <stdlib.h>
#include <cstdlib>
#include <cstdio>

//Funció per a determinar errors
static void HandleError( hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

// MACRO per utilitzar la funció anterior
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void cal_cuda_grises(BYTE * c_greybits, BYTE * c_greybits_PG, BYTE * c_bits, int width, int scan_width)
{
    int i = blockIdx.x;
    int j = blockIdx.y;

    float r = (float) (c_bits[i * scan_width + j * 4 + 0]);
    float g = (float) (c_bits[i * scan_width + j * 4 + 1]);
    float b = (float) (c_bits[i * scan_width + j * 4 + 2]);
    float valor = 0.2126 * r + 0.7152 *g + 0.0722 * b;
    c_greybits[i * width + j] = (BYTE) (valor);
    c_greybits_PG[i * width + j] = c_greybits[i * width + j];              
}

extern "C" void cuda_grises(BYTE * greybits, BYTE * greybits_PG, BYTE * bits, int N, int width, int scan_width)
{
    BYTE * c_greybits;
    BYTE * c_greybits_PG;
    BYTE * c_bits;
    HANDLE_ERROR(hipMalloc((void**)&c_greybits, N*sizeof(BYTE)));
    HANDLE_ERROR(hipMalloc((void**)&c_greybits_PG, N*sizeof(BYTE)));
    HANDLE_ERROR(hipMalloc((void**)&c_bits, N*sizeof(BYTE)));

    HANDLE_ERROR(hipMemcpy(c_greybits, greybits, N*sizeof(BYTE), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(c_greybits_PG, greybits_PG, N*sizeof(BYTE), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(c_bits, bits, N*sizeof(BYTE), hipMemcpyHostToDevice));
    
    cal_cuda_grises<<<N, 1>>>(c_greybits, c_greybits_PG, c_bits, width, scan_width);

    HANDLE_ERROR(hipMemcpy(greybits, c_greybits, N*sizeof(BYTE), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(greybits_PG, c_greybits_PG, N*sizeof(BYTE), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(bits, c_bits, N*sizeof(BYTE), hipMemcpyDeviceToHost));
    
}
